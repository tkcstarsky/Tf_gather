#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <math.h>
#include <ctime>
#include <time.h>

#include <hip/hip_runtime.h>
#include <thrust/host_vector.h> 
#include <thrust/device_vector.h>

using namespace std;

#define N 1         // gather_Nd(N=1 now)
#define L 100       // tensor length
#define I 10        // indices length



int main()
{
    srand(time(0));

    thrust::host_vector<int> H(L); 
    for(int i=0; i<N; i++)
        for(int j=0; j<L)
}